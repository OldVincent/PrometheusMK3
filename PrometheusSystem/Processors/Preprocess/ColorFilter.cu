#include "hip/hip_runtime.h"
#include <opencv4/opencv2/core/cuda_stream_accessor.hpp>

/**
 * @brief 在GPU上运行的颜色筛选器
 * @param min_hue 色调最小值
 * @param max_hue 色调最大值
 * @param min_saturation 饱和度最小值
 * @param max_saturation 饱和度最大值
 * @param min_value 亮度最小值
 * @param max_value 亮度最大值
 * @param source 输入图像
 * @param target 目标二值图
 * @param width 输入图像的宽度
 * @param height 输入图像的高度
 */
__global__ void GPUFilterColor(
		int min_hue, int max_hue,
		int min_saturation, int max_saturation,
		int min_value, int max_value,
		cv::cuda::PtrStepSz<uchar3> source, cv::cuda::PtrStepSz<uchar1> target)
{
	const int id_x = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
	const int id_y = static_cast<int>(blockIdx.y * blockDim.y + threadIdx.y);

	if (id_x >= source.cols || id_y >= source.rows)
	{
		return;
	}

	uchar3 hsv = source(id_y, id_x);

	// 色域条件
	int hue_condition = (hsv.x  - min_hue) * (max_hue - hsv.x);
	// 饱和度条件
	int saturation_condition = (hsv.y - min_saturation) * (max_saturation - hsv.y);
	// 亮度条件
	int value_condition = (hsv.z - min_value) * (max_value - hsv.z);

	// 阈值化设置
	if (hue_condition >= 0 && value_condition >= 0 && saturation_condition >= 0)
	{
		target(id_y, id_x) = make_uchar1(255);
	}
	else
	{

		target(id_y, id_x) = make_uchar1(0);
	}
}

/**
 * @brief 过滤颜色
 * @param min_hue 色调最小值
 * @param max_hue 色调最大值
 * @parma min_saturation 饱和度最小值
 * @param max_saturation 饱和度最大值
 * @param min_value 亮度最小值
 * @param max_value 亮度最大值
 */
extern "C" void FilterColor(
		int min_hue, int max_hue,
		int min_saturation, int max_saturation,
		int min_value, int max_value,
		cv::cuda::GpuMat& source, cv::cuda::GpuMat& target, cv::cuda::Stream& stream)
{
	target = cv::cuda::GpuMat(source.size(), CV_8UC1, cv::Scalar(0,0,0));

	// 图形宽度
	const auto width = source.cols;
	// 图形高度
	const auto height = source.rows;

	// 每个Block要求16x16个线程
	dim3 threadsSize;
	threadsSize.x = 16;
	threadsSize.y = 16;

	// 根据图片大小计算Block个数
	dim3 blocksSize;
	blocksSize.x = width % threadsSize.x == 0 ? width / threadsSize.x : width / threadsSize.x + 1;
	blocksSize.y = height % threadsSize.y == 0 ? height / threadsSize.y : height / threadsSize.y + 1;

	GPUFilterColor<<<blocksSize, threadsSize, 0,  cv::cuda::StreamAccessor::getStream(stream)>>>(
			min_hue, max_hue, min_saturation, max_saturation, min_value, max_value,
			source, target);
}